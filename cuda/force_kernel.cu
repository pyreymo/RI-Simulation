#include "hip/hip_runtime.h"
// force_kernel.cu
#include "force_kernel.cuh"
#include <hip/hip_runtime.h>
#include <stdexcept>

#define CUDA_CHECK(call)                                       \
    do                                                         \
    {                                                          \
        hipError_t err = call;                                \
        if (err != hipSuccess)                                \
        {                                                      \
            throw std::runtime_error(hipGetErrorString(err)); \
        }                                                      \
    } while (0)

__device__ double periodic(double x, double box_length)
{
    return x - floor(x / box_length) * box_length;
}

__global__ void computeForcesKernel(
    const double *x, const double *y,
    double *fx, double *fy,
    int N, double BOX_X, double BOX_Y,
    double CUTOFF_2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N)
        return;

    double xi = x[i];
    double yi = y[i];

    // 避免使用局部变量进行原子操作
    for (int j = 0; j < N; ++j)
    {
        if (i >= j)
            continue; // 避免重复计算

        double dx = xi - x[j];
        double dy = yi - y[j];

        // 应用周期性边界条件
        dx = periodic(dx, BOX_X);
        dy = periodic(dy, BOX_Y);

        double r2 = dx * dx + dy * dy;
        if (r2 < CUTOFF_2)
        {
            double r2i = 1.0 / r2;
            double r6i = r2i * r2i * r2i;
            double lj = 48.0 * r2i * r6i * (r6i - 0.5);
            double fx_ij = lj * dx;
            double fy_ij = lj * dy;

            // 使用原子操作更新全局内存中的力
            atomicAdd(&fx[i], fx_ij);
            atomicAdd(&fy[i], fy_ij);
            atomicAdd(&fx[j], -fx_ij);
            atomicAdd(&fy[j], -fy_ij);
        }
    }
}

// 封装计算力的函数
void computeForces(
    const double *x, const double *y,
    double *fx, double *fy,
    int N, double BOX_X, double BOX_Y, double CUTOFF_2)
{
    // 定义CUDA网格和块的尺寸
    dim3 threadsPerBlock(256);                                           // 每个线程块256个线程
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x); // 计算网格大小

    // 启动CUDA核函数
    computeForcesKernel<<<blocksPerGrid, threadsPerBlock>>>(x, y, fx, fy, N, BOX_X, BOX_Y, CUTOFF_2);

    // 检查内核启动是否有错误
    CUDA_CHECK(hipGetLastError());
}
